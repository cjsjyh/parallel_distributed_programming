#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL(hipEventDestroy(cuda_timer_start));
    CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define BLOCK_SIZE 64
#define DATA_SIZE 10000

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N;
int numbers[DATA_SIZE];
int numbers_CPU[DATA_SIZE];

__global__ void reduction_GPU(int* data, int data_size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < data_size) {
        for (int stride = 1; stride < data_size; stride *= 2) {
            if (idx % (2 * stride) == 0) {
                int lhs = data[idx];
                int rhs = data[idx + stride];
                data[idx] = lhs < rhs ? rhs : lhs;
            }
            __syncthreads();
        }
    }
}

__global__ void reduction_no_diverge_GPU(int* data, int data_size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < data_size) {
        for (int stride = 1; stride < data_size; stride *= 2) {
            int index = 2 * stride * index;
            if (index < blockDim.x) {
                int lhs = data[idx];
                int rhs = data[idx + stride];
                data[idx] = lhs < rhs ? rhs : lhs;
            }
            __syncthreads();
        }
    }
}

#define MAX_CUDA_THREADS_PER_BLOCK 1024
__global__ void reduction_shared_no_diverge_GPU(int* data, int data_size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ float sdata[MAX_CUDA_THREADS_PER_BLOCK];
    if (idx < data_size) {

        /*copy to shared memory*/
        sdata[threadIdx.x] = data[idx];
        __syncthreads();

        for (int stride = 1; stride < blockDim.x; stride *= 2) {
            int index = 2 * stride * index;
            if (index < blockDim.x) {
                int lhs = sdata[threadIdx.x];
                int rhs = sdata[threadIdx.x + stride];
                sdata[threadIdx.x] = lhs < rhs ? rhs : lhs;
            }
            __syncthreads();
        }
    }
    if (idx == 0) data[0] = sdata[0];
}


int reduction_CPU() {
    for (int i = 1; i < DATA_SIZE; i *= 2) {
        for (int j = 0; j < DATA_SIZE; j += i*2) {
            if (j >= DATA_SIZE || j + i >= DATA_SIZE)
                break;
            if (numbers_CPU[j + i] > numbers_CPU[j])
                numbers_CPU[j] = numbers_CPU[j + i];
        }
    }
    //printf("CPU: %d\n", numbers_CPU[0]);
    return numbers_CPU[0];
}


void init_matrix() {
    srand((unsigned)time(NULL));
    int random_num;
    for (int i = 0; i < DATA_SIZE; i++) {
        random_num = (int)(((double)rand() / RAND_MAX) * 300.0f);
        numbers_CPU[i] = numbers[i] = random_num;
    }
}

hipError_t reduction_GPU()
{
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    int max_value;
    int* d_numbers;
    size_t size = DATA_SIZE * sizeof(int);
    CUDA_CALL(hipMalloc(&d_numbers, size));
    
    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(DATA_SIZE / dimBlock.x + 1);


    // with path-divergence
    CUDA_CALL(hipMemcpy(d_numbers, numbers, size, hipMemcpyHostToDevice));

    CHECK_TIME_INIT_GPU();
    CHECK_TIME_START_GPU();

    
    reduction_GPU << < dimGrid, dimBlock >> > (d_numbers, DATA_SIZE);


    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());
    
    CHECK_TIME_END_GPU(device_time);
    CHECK_TIME_DEST_GPU();
    
    CUDA_CALL(hipMemcpy(&max_value, d_numbers, sizeof(int), hipMemcpyDeviceToHost));
    //printf("\nGPU %d\n", max_value);
    printf("GPU time = %.6f\n", device_time);


    // without path-divergence
    CUDA_CALL(hipMemcpy(d_numbers, numbers, size, hipMemcpyHostToDevice));

    CHECK_TIME_INIT_GPU();
    CHECK_TIME_START_GPU();

    reduction_no_diverge_GPU << < dimGrid, dimBlock >> > (d_numbers, DATA_SIZE);

    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());

    CHECK_TIME_END_GPU(device_time);
    CHECK_TIME_DEST_GPU();

    CUDA_CALL(hipMemcpy(&max_value, d_numbers, sizeof(int), hipMemcpyDeviceToHost));
    //printf("\nGPU no divergence: %d\n", max_value);
    printf("GPU no divergence time = %.6f\n", device_time);

    // shared without path-divergence
    CUDA_CALL(hipMemcpy(d_numbers, numbers, size, hipMemcpyHostToDevice));

    CHECK_TIME_INIT_GPU();
    CHECK_TIME_START_GPU();

    reduction_shared_no_diverge_GPU << < dimGrid, dimBlock >> > (d_numbers, DATA_SIZE);

    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());

    CHECK_TIME_END_GPU(device_time);
    CHECK_TIME_DEST_GPU();

    CUDA_CALL(hipMemcpy(&max_value, d_numbers, sizeof(int), hipMemcpyDeviceToHost));
    //printf("\nGPU shared no divergence: %d\n", max_value);
    printf("GPU shared no divergence time = %.6f\n", device_time);


    hipFree(d_numbers);

    
    return cudaStatus;
}

int main()
{
    init_matrix();
    CHECK_TIME_START;
    reduction_CPU();
    CHECK_TIME_END(compute_time);
    printf("CPU time = %.6f\n", compute_time);
    reduction_GPU();

    return 0;
}
