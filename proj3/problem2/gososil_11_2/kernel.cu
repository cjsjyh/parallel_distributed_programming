#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdio.h>
#include<stdlib.h>
#include <math.h>
#include <Windows.h>
#include <time.h>
#include <assert.h>

#define CUDA_CALL(x) { const hipError_t a = (x); if(a != hipSuccess) { printf("\nCuda Error: %s (err_num=%d) at line:%d\n", hipGetErrorString(a), a, __LINE__); hipDeviceReset(); assert(0);}}
typedef float TIMER_T;

#define USE_CPU_TIMER 1
#define USE_GPU_TIMER 1

#if USE_CPU_TIMER == 1
__int64 start, freq, end;
#define CHECK_TIME_START { QueryPerformanceFrequency((LARGE_INTEGER*)&freq); QueryPerformanceCounter((LARGE_INTEGER*)&start); }
#define CHECK_TIME_END(a) { QueryPerformanceCounter((LARGE_INTEGER*)&end); a = (float)((float)(end - start) / (freq / 1000.0f)); }
#else
#define CHECK_TIME_START
#define CHECK_TIME_END(a)
#endif

#if USE_GPU_TIMER == 1
hipEvent_t cuda_timer_start, cuda_timer_stop;
#define CUDA_STREAM_0 (0)

void create_device_timer()
{
    CUDA_CALL(hipEventCreate(&cuda_timer_start));
    CUDA_CALL(hipEventCreate(&cuda_timer_stop));
}

void destroy_device_timer()
{
    CUDA_CALL(hipEventDestroy(cuda_timer_start));
    CUDA_CALL(hipEventDestroy(cuda_timer_stop));
}

inline void start_device_timer()
{
    hipEventRecord(cuda_timer_start, CUDA_STREAM_0);
}

inline TIMER_T stop_device_timer()
{
    TIMER_T ms;
    hipEventRecord(cuda_timer_stop, CUDA_STREAM_0);
    hipEventSynchronize(cuda_timer_stop);

    hipEventElapsedTime(&ms, cuda_timer_start, cuda_timer_stop);
    return ms;
}

#define CHECK_TIME_INIT_GPU() { create_device_timer(); }
#define CHECK_TIME_START_GPU() { start_device_timer(); }
#define CHECK_TIME_END_GPU(a) { a = stop_device_timer(); }
#define CHECK_TIME_DEST_GPU() { destroy_device_timer(); }
#else
#define CHECK_TIME_INIT_GPU()
#define CHECK_TIME_START_GPU()
#define CHECK_TIME_END_GPU(a)
#define CHECK_TIME_DEST_GPU()
#endif

#define BLOCK_SIZE 64
#define TILE_SIZE 32
#define DATA_SIZE (4096 * 4096)
#define TILE_DIM 32

TIMER_T compute_time = 0;
TIMER_T device_time = 0;

int N;
float matrixA[4096 * 4096];
float matrixB[4096 * 4096];
float matrixC[4096 * 4096];



__global__ void matrixMultiplicationKernel(float* A, float* B, float* C) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    float tmpSum = 0;

    if (Row < 4096 && Col < 4096) {
        // each thread computes one element of the block sub-matrix
        for (int i = 0; i < 4096; i++) {
            tmpSum += A[Row * 4096 + i] * B[i * 4096 + Col];
        }
    }
    C[Row * 4096 + Col] = tmpSum;
}

__global__ void matrixMultiplicationSharedKernel(float* A, float* B, float* C)
{
    __shared__ float sA[TILE_SIZE][TILE_SIZE];   // Tile size of 32x32 
    __shared__ float sB[TILE_SIZE][TILE_SIZE];

    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;
    sA[threadIdx.y][threadIdx.x] = 0.0;
    sB[threadIdx.y][threadIdx.x] = 0.0;

    for (int k = 0; k < (((4096 - 1) / TILE_SIZE) + 1); k++)
    {
        if ((Row < 4096) && (threadIdx.x + (k * TILE_SIZE)) < 4096){
            sA[threadIdx.y][threadIdx.x] = A[(Row * 4096) + threadIdx.x + (k * TILE_SIZE)];
        }
        else{
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }
        if (Col < 4096 && (threadIdx.y + k * TILE_SIZE) < 4096){
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k * TILE_SIZE) * 4096 + Col];
        }
        else{
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();

        for (int j = 0; j < TILE_SIZE; ++j){
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }
    }
    if (Row < 4096 && Col < 4096){
        C[Row * 4096 + Col] = Cvalue;
    }
}

void init_matrix() {
    srand((unsigned)time(NULL));
    int random_num;
    for (int i = 0; i < 4096; i++) {
        for (int j = 0; j < 4096; j++) {
            random_num = (int)(((double)rand() / RAND_MAX) * 40.0f);
            matrixA[i*4096 + j] = random_num;
        }
    }

    for (int i = 0; i < 4096; i++) {
        for (int j = 0; j < 4096; j++) {
            random_num = (int)(((double)rand() / RAND_MAX) * 40.0f);
            matrixB[i * 4096 + j] = random_num;
        }
    }
}

hipError_t MatrixMult_GPU()
{
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    float* GPU_matrixA, *GPU_matrixB, *GPU_matrixC;
    size_t size;
    size = DATA_SIZE * sizeof(int);
    CUDA_CALL(hipMalloc(&GPU_matrixA, size));
    CUDA_CALL(hipMalloc(&GPU_matrixB, size));
    CUDA_CALL(hipMalloc(&GPU_matrixC, size));
    CUDA_CALL(hipMemcpy(GPU_matrixA, matrixA, size, hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(GPU_matrixB, matrixB, size, hipMemcpyHostToDevice));
    
    // Global memory
    CHECK_TIME_INIT_GPU();
    CHECK_TIME_START_GPU();

    dim3 dimBlock(BLOCK_SIZE);
    dim3 dimGrid(DATA_SIZE / dimBlock.x);
    matrixMultiplicationKernel << < dimGrid, dimBlock >> > (GPU_matrixA, GPU_matrixB, GPU_matrixC);

    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());
    
    CHECK_TIME_END_GPU(device_time);
    CHECK_TIME_DEST_GPU();
    
    CUDA_CALL(hipMemcpy(matrixC, GPU_matrixC, size, hipMemcpyDeviceToHost));
    printf("Global GPU time = %.6f\n", device_time);


    // Shared memory
    CHECK_TIME_INIT_GPU();
    CHECK_TIME_START_GPU();

    dim3 dimBlockShared(TILE_SIZE, TILE_SIZE);
    dim3 dimGridShared(DATA_SIZE / 4096 / dimBlock.x, DATA_SIZE / 4096 / dimBlock.x);
    matrixMultiplicationSharedKernel << < dimGridShared, dimBlockShared >> > (GPU_matrixA, GPU_matrixB, GPU_matrixC);

    CUDA_CALL(hipGetLastError());
    CUDA_CALL(hipDeviceSynchronize());

    CHECK_TIME_END_GPU(device_time);
    CHECK_TIME_DEST_GPU();

    CUDA_CALL(hipMemcpy(matrixC, GPU_matrixC, size, hipMemcpyDeviceToHost));
    printf("Shared GPU time = %.6f\n", device_time);


    hipFree(GPU_matrixA);
    hipFree(GPU_matrixB);
    hipFree(GPU_matrixC);

    
    return cudaStatus;
}

int main()
{
    init_matrix();
    MatrixMult_GPU();

    return 0;
}
